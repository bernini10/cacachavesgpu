#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "sha256_cuda.h"
#include "ripemd160_cuda.h"

// Kernel to compute Bitcoin addresses in parallel
__global__ void bitcoin_address_kernel(const unsigned char* public_keys, unsigned char* bitcoin_addresses, const unsigned char* target_address, int* match_found, int num_keys) {
    extern __shared__ unsigned char shared_memory[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_keys) {
        if (*match_found) return;

        unsigned char* sha256_hash = &shared_memory[threadIdx.x * SHA256_DIGEST_SIZE];
        unsigned char* ripemd160_hash = &shared_memory[blockDim.x * SHA256_DIGEST_SIZE + threadIdx.x * RIPEMD160_DIGEST_SIZE];

        // Step 1: Apply SHA-256 to the public key
        sha256_gpu(&public_keys[idx * 33], 33, sha256_hash);

        // Step 2: Apply RIPEMD-160 to the SHA-256 hash
        ripemd160_gpu(sha256_hash, SHA256_DIGEST_SIZE, ripemd160_hash);

        // Step 3: Copy the result to the global memory
        memcpy(&bitcoin_addresses[idx * RIPEMD160_DIGEST_SIZE], ripemd160_hash, RIPEMD160_DIGEST_SIZE);

        // Step 4: Check for a match
        if (memcmp(ripemd160_hash, target_address, RIPEMD160_DIGEST_SIZE) == 0) {
            atomicExch(match_found, 1);
        }
    }
}
